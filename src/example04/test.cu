#include "hip/hip_runtime.h"
﻿#include <iostream>

#include "test.cuh"

// 线程(Thread)：一般通过GPU的一个核进行处理;
// 线程块(Block)：由多个线程组成；各block是并行执行的，block间无法通信，也没有执行顺序。
// 线程格(Grid)：由多个线程块组成。
// 核函数(Kernel)：在GPU上执行的函数通常称为核函数;一般通过标识符__global__修饰，
// 调用通过<<<参数1,参数2>>>，用于说明内核函数中的线程数量，以及线程是如何组织的。
// gridDim：gridDim.x、gridDim.y、gridDim.z分别表示线程格各个维度的大小
// blockDim：blockDim.x、blockDim.y、blockDim.z分别表示线程块中各个维度的大小
// blockIdx：blockIdx.x、blockIdx.y、blockIdx.z分别表示当前线程块所处的线程格的坐标位置
// threadIdx：threadIdx.x、threadIdx.y、threadIdx.z分别表示当前线程所处的线程块的坐标位置
// 线程格里面总的线程个数N：N = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z

//定义核函数 __global__为声明关键字
template<typename T>
__global__ void matAdd_cuda(T *a, T *b, T *sum)
{
    // blockIdx代表block的索引, blockDim代表block的大小，threadIdx代表thread线程的索引，
    // 因此对于一维的block和thread索引的计算方式如下
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sum[i] = a[i] + b[i];
}

// 核函数用模板不会报错，模板名字是具有链接的，但它们不能具有C链接，因此不能用在供调用的函数上
void matAdd(float *a, float *b, float * sum, int length)
{
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount( &count );
    for (int i = 0; i < count; i++) {

        hipGetDeviceProperties(&prop, i);

        // multiProcessorCount: 设备上的流多处理器（SM）的数量
        // maxThreadsPerMultiProcessor: 每个流多处理器（SM）最大线程数量
        std::cout << prop.multiProcessorCount << " " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << prop.maxBlocksPerMultiProcessor << " " << prop.maxThreadsPerBlock << std::endl;
        std::cout << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << std::endl;
    }

    // 设置使用第0块GPU进行运算，并且设置运算显卡
    int device = 0;
    hipSetDevice(device);

    // 获取对应设备属性
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    // 每个线程块的最大线程数
    int threadMaxSize = devProp.maxThreadsPerBlock;

    // 计算Block大小,block一维度是最大的，一般不会溢出
    int blockSize = (length + threadMaxSize - 1) / threadMaxSize;

    // 设置thread
    dim3 thread(threadMaxSize);

    // 设置block
    dim3 block(blockSize);

    // 计算空间大小
    int size = length * sizeof(float);

    float *sumGPU = nullptr, *aGPU = nullptr, *bGPU = nullptr;

    // 开辟显存空间
    hipMalloc((void **) &sumGPU, size);
    hipMalloc((void **) &aGPU, size);
    hipMalloc((void **) &bGPU, size);

    // 内存->显存
    hipMemcpy((void *) aGPU, (void *) a, size, hipMemcpyHostToDevice);
    hipMemcpy((void *) bGPU, (void *) b, size, hipMemcpyHostToDevice);

    // 运算
    matAdd_cuda<float><<<block, thread>>>(aGPU, bGPU, sumGPU);

    // hipDeviceSynchronize();

    // 显存->内存
    hipMemcpy(sum, sumGPU, size, hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(sumGPU);
    hipFree(aGPU);
    hipFree(bGPU);
}
